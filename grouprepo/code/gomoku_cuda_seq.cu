#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <set>
#include <bits/stdc++.h>
#include <hip/hip_runtime.h>
#include "common.hpp"
#include <hip/hip_runtime_api.h>
#include <stdio.h>
using namespace std;

// Dimension of board
#define dimension 9

// Define the total depth and the sequential depth to be searched
#define totalDepth 3
#define sequentialDepth 2

// Structs
typedef struct {
	int x;
	int y;
} move_t;

typedef struct {
	int score;
	bool gameOver;
} score_t;

typedef struct {
	int movesNumOne;
	int movesNumTwo;
	move_t* movesOne;
	move_t* movesTwo;
} playedMoves_t;


class Board {
public:
int board[dimension * dimension];         // the board 0 for empty, 1 for black and 2 for white
__host__ __device__ Board(){         // constructor
	for (int i = 0; i < dimension; i++) {
		for (int j = 0; j < dimension; j++) {
			board[i * dimension + j] = 0;
		}
	}
}

__host__ __device__ Board(const Board &toBeCopied){         // Copy constructor
	for (int i = 0; i < dimension * dimension; i++) {
		board[i] = toBeCopied.board[i];
	}
}


/*
 * Helper function to print the board status
 */
__host__ __device__ void printBoard() {
	for (int i = 0; i < dimension; i++) {
		printf("%d: ", i);
		for (int j = 0; j < dimension; j++) {
			printf("%d ", board[i * dimension + j]);
		}
		printf("\n");
	}
	printf("\n");
}

/*
 * Add a stone of 'color' to the location
 */
__host__ __device__ void addStone(int color, int location_x, int location_y){
	board[location_x * dimension + location_y] = color;
}

/*
 * Remove the stone at given location
 */
 __host__ __device__  void removeStone(int location_x, int location_y){
	board[location_x * dimension + location_y] = 0;
}

/*
 * Getter
 */
__host__ __device__ int getElement(int location_x, int location_y){
	return board[location_x * dimension + location_y];
}

/*
 * Setter
 */
__host__ __device__ void setBoard(int tempBoard[]){
	for (int i = 0; i < dimension*dimension; i++){
		board[i] = tempBoard[i];
	}
}

/*
 * Get the number of possible moves at this board state
 */
__host__ __device__ int getPossibleMovesCount(){
	int sum = 0;
	for (int i = 0; i < dimension; i++){
		for (int j = 0; j < dimension; j++){
			if (board[i * dimension + j] == 0){
				sum++;
			}
		}
	}
	return sum;
}
/*
 * Get a list of current possible moves. Possible moves are defined to be those positions (i,j) such that any adjacent cells, including diagonals have a stone placed
 */
 __host__ __device__ move_t* getPossibleMoves(playedMoves_t played, int* sum){
	Board tempBoard;
	tempBoard.setBoard(board);
	for (int oneMoves = 0; oneMoves < played.movesNumOne; oneMoves++){
		tempBoard.addStone(1, played.movesOne[oneMoves].x, played.movesOne[oneMoves].y);
	}
	for (int twoMoves = 0; twoMoves < played.movesNumTwo; twoMoves++){
		tempBoard.addStone(2, played.movesTwo[twoMoves].x, played.movesTwo[twoMoves].y);
	}
	int moveCount = tempBoard.getPossibleMovesCount();
	*sum = moveCount;
	move_t* moves = (move_t*) malloc(sizeof(move_t) * tempBoard.getPossibleMovesCount());
	int index = 0;

	for (int i = 0; i < dimension; i++){
		for (int j = 0; j < dimension; j++){
			if (tempBoard.getElement(i, j) == 0){
				moves[index].x = i;
				moves[index].y = j;
				index++;
			}
		}
	}
	
	return moves;
}


/*
 * Function that gives a score based on the number of consecutive stones, whether they have open ends and whose turn it currently is. To be called by the other heuristic functions to calculate score
 */
 __host__ __device__ int shapeScore(int countConsecutive, int openEnds, bool playersTurn){


	if (openEnds == 0 && countConsecutive < 5) {
		return 0;
	}
	switch(countConsecutive) {
	case 4:
		switch (openEnds) {
		case 1:
			if (playersTurn) {
				return 100000000;
			} else{
				return 50;
			}
		case 2:
			if (playersTurn) {
				return 100000000;
			} else{
				return 500000;
			}
		}
	case 3:
		switch (openEnds) {
		case 1:
			if (playersTurn) {
				return 7;
			} else{
				return 5;
			}
		case 2:
			if (playersTurn) {
				return 10000;
			} else{
				return 50;
			}
		}
	case 2:
		switch (openEnds) {
		case 1:
			return 3;
		case 2:
			return 5;
		}
	case 1:
		switch (openEnds) {
		case 1:
			return 1;
		case 2:
			return 2;
		}
	default:
		return 200000000;
	}
	return 0;
}

/*
 * Using the shapreScore function, calculate the score for the player 'evaluateFor' and at column 'columnNum' given that the current player is 'currentPlayer'
 */
 __host__ __device__ score_t calculateScoreVertical(int currentPlayer, int evaluateFor, int columnNum){
	int score = 0;
	int openEnds = 0;
	int consecutiveCount = 0;
	bool gameOver = false;

	for (int i = 0; i < dimension; i++) {
		if (board[i * dimension + columnNum] == evaluateFor) {        // If the color is the one we are counting, increment
			consecutiveCount++;
		} else if (board[i * dimension + columnNum] == 0 && consecutiveCount > 0) {        // If the cell is empty and theres been more than one consecutive, increment openends and restart counters
			openEnds++;
			score += shapeScore(consecutiveCount, openEnds, evaluateFor == currentPlayer);
			consecutiveCount = 0;
			openEnds = 1;
		} else if (board[i * dimension + columnNum] == 0) {        // If the cell is empty and no consecutive
			openEnds = 1;
		} else if (consecutiveCount > 0) {        //  If there's been more than one consecutive but a dead end
			score += shapeScore(consecutiveCount, openEnds, evaluateFor == currentPlayer);
			consecutiveCount = 0;
			openEnds = 0;
		} else {         // Just the opposite color with no consecutive
			openEnds=0;
		}
		if (consecutiveCount >= 5) {
			gameOver = true;
		}
	}
	if (consecutiveCount > 0) {        // Account for consecutive ending at the last cell
		score += shapeScore(consecutiveCount, openEnds, evaluateFor == currentPlayer);
	}

	score_t ret = {score, gameOver};
	return ret;

}

/*
 * Using the shapreScore function, calculate the score for the player 'evaluateFor' and at row 'rowNum' given that the current player is 'currentPlayer'
 */
 __host__ __device__ score_t calculateScoreHorizontal(int currentPlayer, int evaluateFor, int rowNum){
	int score = 0;
	int openEnds = 0;
	int consecutiveCount = 0;
	bool gameOver = false;

	for (int i = 0; i < dimension; i++) {
		if (board[rowNum * dimension + i] == evaluateFor) {        // If the color is the one we are counting, increment
			consecutiveCount++;
		} else if (board[rowNum * dimension + i] == 0 && consecutiveCount > 0) {        // If the cell is empty and theres been more than one consecutive, increment openends and restart counters
			openEnds++;
			score += shapeScore(consecutiveCount, openEnds, evaluateFor == currentPlayer);
			consecutiveCount = 0;
			openEnds = 1;
		} else if (board[rowNum * dimension + i] == 0) {        // If the cell is empty and no consecutive
			openEnds = 1;
		} else if (consecutiveCount > 0) {        //  If there's been more than one consecutive but a dead end
			score += shapeScore(consecutiveCount, openEnds, evaluateFor == currentPlayer);
			consecutiveCount = 0;
			openEnds = 0;
		} else {         // Just the opposite color with no consecutive
			openEnds=0;
		}
		if (consecutiveCount >= 5) {
			gameOver = true;
		}
	}
	if (consecutiveCount > 0) {        // Account for consecutive ending at the last cell
		score += shapeScore(consecutiveCount, openEnds, evaluateFor == currentPlayer);
	}

	score_t ret = {score, gameOver};
	return ret;
}


/*
 * Using the shapreScore function, calculate the score for the player 'evaluateFor' and at row 'rowNum' given that the current player is 'currentPlayer' at the diagonal
 */
__host__ __device__ score_t calculateScoreDiagonalLR(int currentPlayer, int evaluateFor){

	int score = 0;
	int openEnds = 0;
	int consecutiveCount = 0;
	bool gameOver = false;

	for (int i = 0; i < 2 * dimension - 1; i++) {
		consecutiveCount = 0;
		openEnds = 0;

		int z = (i < dimension) ? 0 : i - dimension + 1;
		for (int j = z; j <= i - z; j++) {
			if (board[j * dimension + (i - j)] == evaluateFor) {        // If the color is the one we are counting, increment
				consecutiveCount++;
			} else if (board[j * dimension + (i - j)] == 0 && consecutiveCount > 0) {        // If the cell is empty and theres been more than one consecutive, increment openends and restart counters
				openEnds++;
				score += shapeScore(consecutiveCount, openEnds, evaluateFor == currentPlayer);
				consecutiveCount = 0;
				openEnds = 1;
			} else if (board[j * dimension + (i - j)] == 0) {        // If the cell is empty and no consecutive
				openEnds = 1;
			} else if (consecutiveCount > 0) {        //  If there's been more than one consecutive but a dead end
				score += shapeScore(consecutiveCount, openEnds, evaluateFor == currentPlayer);
				consecutiveCount = 0;
				openEnds = 0;
			} else {         // Just the opposite color with no consecutive
				openEnds=0;
			}
			if (consecutiveCount >= 5) {
				gameOver = true;
			}
		}
		if (consecutiveCount > 0) {        // Account for consecutive ending at the last cell
			score += shapeScore(consecutiveCount, openEnds, evaluateFor == currentPlayer);
		}
	}

	score_t ret = {score, gameOver};
	return ret;
}

/*
 * Using the shapreScore function, calculate the score for the player 'evaluateFor' and at row 'rowNum' given that the current player is 'currentPlayer' at the diagonal
 */
__host__ __device__ score_t  calculateScoreDiagonalRL(int currentPlayer, int evaluateFor){

	int score = 0;
	int openEnds = 0;
	int consecutiveCount = 0;
	bool gameOver = false;

	for (int i = 0; i < 2 * dimension - 1; i++) {
		consecutiveCount = 0;
		openEnds = 0;

		int z = (i < dimension) ? 0 : i - dimension + 1;
		for (int j = z; j <= i - z; j++) {
			if (board[j * dimension + (dimension-1-i+j)] == evaluateFor) {        // If the color is the one we are counting, increment
				consecutiveCount++;
			} else if (board[j * dimension + (dimension-1-i+j)] == 0 && consecutiveCount > 0) {        // If the cell is empty and theres been more than one consecutive, increment openends and restart counters
				openEnds++;
				score += shapeScore(consecutiveCount, openEnds, evaluateFor == currentPlayer);
				consecutiveCount = 0;
				openEnds = 1;
			} else if (board[j * dimension + (dimension-1-i+j)] == 0) {        // If the cell is empty and no consecutive
				openEnds = 1;
			} else if (consecutiveCount > 0) {        //  If there's been more than one consecutive but a dead end
				score += shapeScore(consecutiveCount, openEnds, evaluateFor == currentPlayer);
				consecutiveCount = 0;
				openEnds = 0;
			} else {         // Just the opposite color with no consecutive
				openEnds=0;
			}
			if (consecutiveCount >= 5) {
				gameOver = true;
			}
		}
		if (consecutiveCount > 0) {        // Account for consecutive ending at the last cell
			score += shapeScore(consecutiveCount, openEnds, evaluateFor == currentPlayer);
		}
	}
	score_t ret = {score, gameOver};
	return ret;
}

/*
 * Calcuate the whole board score for player 'evaluateFor' given that the current player is 'currentPlayer'
 */
 __host__ __device__ score_t calculateBoardScorePlayer(int currentPlayer, int evaluateFor){
	int score = 0;
	bool gameOver = false;
	for (int i = 0; i < dimension; i++) {
		score_t horizontal = calculateScoreHorizontal(currentPlayer, evaluateFor, i);
		score_t vertical = calculateScoreVertical(currentPlayer, evaluateFor, i);
		score += horizontal.score + vertical.score;
		gameOver = gameOver || horizontal.gameOver || vertical.gameOver;
	}
	score_t diagonalLR = calculateScoreDiagonalLR(currentPlayer, evaluateFor);
	score_t diagonalRL = calculateScoreDiagonalRL(currentPlayer, evaluateFor);
	score += diagonalLR.score + diagonalRL.score;
	gameOver = gameOver || diagonalLR.gameOver || diagonalRL.gameOver;
	score_t ret = {score, gameOver};
	return ret;
}
/*
 * Calculate the whole board score for the current player. Defined to be the difference between the score evaluated for the current player and the score evaluated for the opponent
 */
 __host__ __device__ score_t calculateBoardScoreTotal(playedMoves_t played, int currentPlayer){
	Board tempBoard;
	tempBoard.setBoard(board);
    for (int oneMoves = 0; oneMoves < played.movesNumOne; oneMoves++){
	 	tempBoard.addStone(1, played.movesOne[oneMoves].x, played.movesOne[oneMoves].y);
	}
	for (int twoMoves = 0; twoMoves < played.movesNumTwo; twoMoves++){
		tempBoard.addStone(2, played.movesTwo[twoMoves].x, played.movesTwo[twoMoves].y);
	}
	score_t cur = tempBoard.calculateBoardScorePlayer(currentPlayer, currentPlayer);
	score_t other = tempBoard.calculateBoardScorePlayer(currentPlayer, (currentPlayer == 1) ? 2 : 1);
	score_t ret = {cur.score - other.score, cur.gameOver || other.gameOver};
	// score_t ret = {0, false};

	return ret;
}
};

__global__ void minimaxKernel(Board* device_currentBoard, move_t* device_move_array, int n, move_t* device_bestMove, int* score, int depth, bool maximizingPlayer, int currentPlayer);
__global__ void minimaxKernelSeqPar(Board* device_boards, int* device_scores, int leavesCount, int parDepth, int currentPlayer, bool maximizingPlayer);



class Minimax {
private:
Board board;
public:
__host__ __device__ Minimax(Board miniMaxBoard){
	board = miniMaxBoard;
}
	
/*
 * Given some set of old moves, and a new move, make a new playedMoves object, add to that and return
 */
__host__ __device__ playedMoves_t updatePlayedMoves(playedMoves_t oldPlayed, move_t nextMove, int currentPlayer){
	playedMoves_t newPlayed = {oldPlayed.movesNumOne, oldPlayed.movesNumTwo, NULL, NULL};
	// Allocate memory, copy stuff and return
	if (currentPlayer == 1){
		newPlayed.movesNumOne = oldPlayed.movesNumOne+1;
		newPlayed.movesOne = (move_t*) malloc(sizeof(move_t) * newPlayed.movesNumOne);
		newPlayed.movesTwo = (move_t*) malloc(sizeof(move_t) * newPlayed.movesNumTwo);
		for (int i = 0; i < oldPlayed.movesNumOne; i++){
			newPlayed.movesOne[i].x = oldPlayed.movesOne[i].x;
			newPlayed.movesOne[i].y = oldPlayed.movesOne[i].y;
		}
		for (int j = 0; j < oldPlayed.movesNumTwo; j++){
			newPlayed.movesTwo[j].x = oldPlayed.movesTwo[j].x;
			newPlayed.movesTwo[j].y = oldPlayed.movesTwo[j].y;
		}
		newPlayed.movesOne[newPlayed.movesNumOne - 1].x = nextMove.x;
		newPlayed.movesOne[newPlayed.movesNumOne - 1].y = nextMove.y;
	} else {
		newPlayed.movesNumTwo = oldPlayed.movesNumTwo+1;
		newPlayed.movesOne = (move_t*) malloc(sizeof(move_t) * newPlayed.movesNumOne);
		newPlayed.movesTwo = (move_t*) malloc(sizeof(move_t) * newPlayed.movesNumTwo);
		for (int i = 0; i < oldPlayed.movesNumOne; i++){
			newPlayed.movesOne[i].x = oldPlayed.movesOne[i].x;
			newPlayed.movesOne[i].y = oldPlayed.movesOne[i].y;
		}
		for (int j = 0; j < oldPlayed.movesNumTwo; j++){
			newPlayed.movesTwo[j].x = oldPlayed.movesTwo[j].x;
			newPlayed.movesTwo[j].y = oldPlayed.movesTwo[j].y;
		}
		newPlayed.movesTwo[newPlayed.movesNumTwo - 1].x = nextMove.x;
		newPlayed.movesTwo[newPlayed.movesNumTwo - 1].y = nextMove.y;
	}
	return newPlayed;
}


/*
 * Do minimax with alpha-beta pruning from the initial board state with playedMoves_t 'played'
 */
__host__ __device__ int doMinimaxAB(playedMoves_t played, int depth, bool maximizingPlayer, int currentPlayer, int maximizer, int alpha, int beta){
	score_t status = board.calculateBoardScoreTotal(played, maximizer);
	// At a terminal node, return the evaluation
	if (depth == 0 || status.gameOver) {
		return status.score;
	}
	int movesCount;
	move_t* moves = board.getPossibleMoves(played, &movesCount);

	// recursively call minimax for each child move
	if (maximizingPlayer) {
		int maxEval = INT_MIN;
		for (int i = 0; i < movesCount; i++) {
			move_t move = moves[i];
			playedMoves_t newPlayed = updatePlayedMoves(played, move, currentPlayer);
			int eval = doMinimaxAB(newPlayed, depth - 1, false, (currentPlayer == 1) ? 2 : 1, maximizer, alpha, beta);
			maxEval = max(maxEval, eval);
			alpha = max(alpha, eval);
			if (beta <= alpha) {
				break;
			}
			free(newPlayed.movesOne);
			free(newPlayed.movesTwo);
		}
		free(moves);
		return maxEval;
	} else{
		int minEval = INT_MAX;
		for (int i = 0; i < movesCount; i++) {
			move_t move = moves[i];
			playedMoves_t newPlayed = updatePlayedMoves(played, move, currentPlayer);
			int eval = doMinimaxAB(newPlayed, depth - 1, true, (currentPlayer == 1) ? 2 : 1,maximizer, alpha, beta);
			minEval= min(minEval, eval);
			beta = min(beta, eval);
			if (beta <= alpha) {
				break;
			}
			free(newPlayed.movesOne);
			free(newPlayed.movesTwo);
		}
		free(moves);
		return minEval;
	}
}




// Get all the leaf boards at the sequential depth using the recursive helper function
Board* getLeafBoards(int seqDepth, int currentPlayer, int leavesCount){
	Board* ret = (Board*) malloc(sizeof(Board) * leavesCount);
	/* cout << leavesCount << endl; */
	getLeafBoardsHelper(ret, board, seqDepth, currentPlayer, 0);
	return ret;
}

// Recursively add the terminal boards to the 'boards' list
int getLeafBoardsHelper(Board* boards, Board currentBoard, int seqDepth, int currentPlayer, int curIndex){
	if (seqDepth == 0){	
		boards[curIndex] = currentBoard;
		return curIndex+1;
	} else {
		Board tempBoard = currentBoard;
		int movesCount;
		playedMoves_t played = {0, 0, NULL, NULL};
		move_t* moves = tempBoard.getPossibleMoves(played, &movesCount);
		for (int i = 0; i < movesCount; i++){
			move_t move = moves[i];
			tempBoard.addStone(currentPlayer, move.x, move.y);
			curIndex = getLeafBoardsHelper(boards, tempBoard, seqDepth-1, (currentPlayer == 1) ? 2 : 1, curIndex);
			tempBoard.removeStone(move.x, move.y);
		}
		return curIndex;
	}
}

// Gather the result from the GPU computation in the CPU by tracing down the game tree
pair<int, int> getMinimaxSeqParAfterGPU(int* scores, playedMoves_t played, int depth, bool maximizingPlayer, int currentPlayer, int maximizer, int curIndex){
	if (depth == 0) {
		// At a terminal node, instead of calculating the score, just look up on the scores array
		int ret = scores[curIndex];
		return make_pair(ret, curIndex + 1);
	}
	int movesCount;
	move_t* moves = board.getPossibleMoves(played, &movesCount);

	if (maximizingPlayer) {
		int maxEval = INT_MIN;
		for (int i = 0; i < movesCount; i++) {
			move_t move = moves[i];
			playedMoves_t newPlayed = updatePlayedMoves(played, move, currentPlayer);
			pair<int, int> result = getMinimaxSeqParAfterGPU(scores, newPlayed, depth - 1, false, (currentPlayer == 1) ? 2 : 1, maximizer, curIndex);	
			int eval = result.first;
			curIndex = result.second;
			maxEval = max(maxEval, eval);
			free(newPlayed.movesOne);
			free(newPlayed.movesTwo);
		}
		free(moves);
		return make_pair(maxEval, curIndex);
	} else{
		int minEval = INT_MAX;
		for (int i = 0; i < movesCount; i++) {
			move_t move = moves[i];
			playedMoves_t newPlayed = updatePlayedMoves(played, move, currentPlayer);
			pair<int, int> result = getMinimaxSeqParAfterGPU(scores, newPlayed, depth - 1, true, (currentPlayer == 1) ? 2 : 1, maximizer, curIndex);
			int eval = result.first;
			curIndex = result.second;
			minEval= min(minEval, eval);
			free(newPlayed.movesOne);
			free(newPlayed.movesTwo);
		}
		free(moves);
		return make_pair(minEval, curIndex);
	}
}


// Get the best move by gathering the result from the GPU computation in the CPU by tracing down the game tree
move_t retrieveBestMoveAfterGPU(int depth, bool maximizingPlayer, int* host_scores, int currentPlayer){
	
	playedMoves_t played = {0, 0, NULL, NULL};
	int movesCount;
	move_t* moves = board.getPossibleMoves(played, &movesCount);
	printf("Size of options: %d\n", movesCount);
	/* for (int i = 0; i<movesCount; i++) { */
	/* 	cout << moves[i].x << ", " << moves[i].y << endl; */
	/* } */
	int bestVal = INT_MIN;
	move_t bestMove = moves[0];
	int curIndex = 0;
	for (int i = 0; i < movesCount; i++) {
		move_t move = moves[i];
		playedMoves_t newPlayed = updatePlayedMoves(played, move, currentPlayer);
		// Call the getMinimaxSeqParAfterGPU function to retrieve the results
		pair<int, int> result = getMinimaxSeqParAfterGPU(host_scores, newPlayed, depth - 1, !maximizingPlayer, (currentPlayer == 1) ? 2 : 1, currentPlayer, curIndex);
		int eval = result.first;
		curIndex = result.second;
		// cout << curIndex << endl;
		/* printf("%d, %d Score: %d\n", move.x, move.y, eval); */
		if (eval > bestVal) {
			bestVal = eval;
			bestMove = move;
		}
		free(newPlayed.movesOne);
		free(newPlayed.movesTwo);
	}
	printf("Best score: %d\n", bestVal);
	printf("Best move %d : %d\n", bestMove.x, bestMove.y);
	free(moves);
	return bestMove;
}

// On the GPU get the best move using sequential parallel GPU implementation
move_t* getBestMoveABSeqPar(int seqDepth, int depth, bool maximizingPlayer, int currentPlayer){
	int parDepth = depth - seqDepth;
	int curOptions = board.getPossibleMovesCount();
	int leavesCount = curOptions;
	for (int i = 0; i < seqDepth - 1; i++){
		leavesCount *= (curOptions - (i+1));
	}
	/* cout << "leavesCount: " << leavesCount << endl; */
	Board* host_boards = getLeafBoards(seqDepth, currentPlayer, leavesCount);
	int* host_scores = (int*) malloc(sizeof(int) * leavesCount);

	Board* device_boards;
	int* device_scores;
	
	// Copy over the data to device
	/* printf("%ld\n", sizeof(Board)*leavesCount); */
	if (hipMalloc(&device_boards, sizeof(Board) * leavesCount) != hipSuccess) {
		fprintf(stderr, "Error: %s hipMalloc at line %d in function %s\n", hipGetErrorString(hipGetLastError()), (__LINE__), (__func__));
	}
	hipError_t err = hipMemcpy(device_boards, host_boards, sizeof(Board)*leavesCount, hipMemcpyHostToDevice);
	if (err != hipSuccess){
		printf("Error: %s hipMemcpy at line %d\n", hipGetErrorString(err), (__LINE__));
	}

	// checkCudaErrors(hipMalloc((void**) &device_scores, sizeof(int) * leavesCount));
	if (hipMalloc(&device_scores, sizeof(int) * leavesCount) != hipSuccess) {
		fprintf(stderr, "Error: hipMalloc at line %d in function %s\n", (__LINE__), (__func__));
	}

	int currentPlayerGPU = currentPlayer;
	if (seqDepth % 2 == 1){
		currentPlayerGPU = (currentPlayer == 1) ? 2 : 1;
	}
	// Declare thread block size etc. and launch threads
	int num_threads_per_block = 128;
	int num_blocks =  (leavesCount + num_threads_per_block - 1) / num_threads_per_block;
	minimaxKernelSeqPar<<<num_blocks, num_threads_per_block>>>(device_boards, device_scores, leavesCount, parDepth, currentPlayerGPU, seqDepth % 2 == 0);
	// hipDeviceSynchronize();
	err = hipGetLastError();
	if (err != hipSuccess){
		printf("Error: %s kernel launch at line %d\n", hipGetErrorString(err), (__LINE__));
	}
	move_t* host_bestMove = (move_t*) malloc(sizeof(move_t));
	// get_result_gpu(device_scores, host_scores, leavesCount);
	err = hipMemcpy(host_scores, device_scores, sizeof(int) * leavesCount, hipMemcpyDeviceToHost);
	if (err != hipSuccess){
		printf("Error: %s hipMemcpy launch at line %d\n", hipGetErrorString(err), (__LINE__));
	}
	*host_bestMove =  retrieveBestMoveAfterGPU(seqDepth, maximizingPlayer, host_scores, currentPlayer);
	free(host_boards);
	free(host_scores);
	hipFree(device_boards);
	hipFree(device_scores);
	return host_bestMove;
}
};

// GPU kernel to do parallel minimax search
__global__ void minimaxKernelSeqPar(Board* device_boards, int* device_scores, int leavesCount, int parDepth, int currentPlayer, bool maximizingPlayer){
	int tid  = blockDim.x * blockIdx.x + threadIdx.x;

	if (tid < leavesCount){
		Minimax minimax(device_boards[tid]);
		playedMoves_t played = {0, 0, NULL, NULL};
		int maximizer = currentPlayer;
		if (!maximizingPlayer){ 
			maximizer = (currentPlayer == 1) ? 2 : 1;
		}
		int score = minimax.doMinimaxAB(played, parDepth, maximizingPlayer, currentPlayer, maximizer, INT_MIN, INT_MAX);
		// int score = 0;
		device_scores[tid] = score;
		// printf("%d : Score: %d\n", tid, score);
	}
}



int main() {

	hipDeviceSetLimit(hipLimitStackSize, 30000);
	hipDeviceSetLimit(hipLimitMallocHeapSize, 100000000);
	size_t free, total;
	hipMemGetInfo(&free, &total);
	/* printf("%ld, %ld\n", free, total); */
	
	Board board;
	board.addStone(1, 0, 0);
	// board.addStone(1, 2,3); 
	// board.addStone(1, 1,4); 
	// board.addStone(2, 0,4); 
	// board.addStone(1, 4,4); 
	// board.addStone(2, 4,0); 
	// board.addStone(2, 4, 1); 
	board.printBoard();

	uint64_t start_t;
	uint64_t end_t;
	InitTSC();


	Minimax minimax(board);
	start_t = ReadTSC();
	move_t* device_bestMove = minimax.getBestMoveABSeqPar(sequentialDepth, totalDepth, true, 2);
	end_t = ReadTSC();
	board.printBoard();
	printf("Time to run Minimax is %g\n", ElapsedTime(end_t - start_t));
	cout << "Best move: " << (*device_bestMove).x << ", " << (*device_bestMove).y << endl;

    	/* free(device_bestMove); */
	return 0;
}
